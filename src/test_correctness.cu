#include "hip/hip_runtime.h"
#include <cmath>
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "common.cuh"

// Oracle CPU implementation of SGEMM.
void sgemm_oracle(
    const uint32_t M,
    const uint32_t N,
    const uint32_t K,
    const float alpha,
    const float *A,
    const float *B,
    const float beta,
    float *C
) {
    for (uint32_t i = 0; i < M; ++i) {
        for (uint32_t j = 0; j < N; ++j) {
            float accumulate = 0.0f;
            for (uint32_t k = 0; k < K; ++k) {
                accumulate += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = alpha * accumulate + beta * C[i * N + j];
        }
    }
}

// Checks for matrix equality within atol and rtol.
bool check_matrix_equality(
    const uint32_t M,
    const uint32_t N,
    float *A_oracle,
    float *A
) {
    const float atol = 1e-4f;
    const float rtol = 1e-3f;

    for (uint32_t i = 0; i < M * N; ++i) {
        const float lhs = std::fabs(A_oracle[i] - A[i]);
        const float rhs = atol + rtol * std::fabs(A_oracle[i]);
        if (lhs > rhs) {
            return false;
        }
    }

    return true;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Please specify the version: "
                  << argv[0] << " <version>" << std::endl;
        return 1;
    }
    const int sgemm_version = std::atoi(argv[1]);
    if (sgemm_version < 0 || sgemm_version > NUM_SGEMM_VERSIONS) {
        std::cerr << "Version must be between 0 and " << NUM_SGEMM_VERSIONS
                 << std::endl;
        return 1;
    }

    const uint32_t M = 1024, N = 1024, K = 1024;
    const float alpha = 1.0f, beta = 0.0f;

    // Generating cuBLAS handle in case we want to run cuBLAS.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *A, *B, *C, *C_oracle;
    float *A_device, *B_device, *C_device;

    A = new float[M * K];
    B = new float[K * N];
    C = new float[M * N];
    C_oracle = new float[M * N];

    init_matrix(M, K, A);
    init_matrix(K, N, B);
    init_matrix(M, N, C);
    std::memcpy(C_oracle, C, sizeof(float) * M * N);
    std::cout << "Generated matrices." << std::endl;

    // Run oracle.
    std::cout << "Oracle SGEMM started." << std::endl;
    sgemm_oracle(M, N, K, alpha, A, B, beta, C_oracle);
    std::cout << "Oracle SGEMM completed." << std::endl;

    // Run CUDA.
    std::cout << "CUDA SGEMM started." << std::endl;
    hipMalloc(&A_device, M * K * sizeof(float));
    hipMalloc(&B_device, K * N * sizeof(float));
    hipMalloc(&C_device, M * N * sizeof(float));

    hipMemcpy(A_device, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C_device, C, M * N * sizeof(float), hipMemcpyHostToDevice);

    if (sgemm_version == 0) {
        // If sgemm_version == 0 we call cuBLAS.
        v0::sgemm(
            M, N, K, alpha, A_device, B_device, beta, C_device, handle);
    } else {
        // Get the chosen SGEMM function version.
        SGEMMFunc sgemm_func = SGEMM_FUNCS[sgemm_version - 1];
        sgemm_func(M, N, K, alpha, A_device, B_device, beta, C_device);
    }

    hipMemcpy(C, C_device, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "CUDA SGEMM completed." << std::endl;

    // Check for correctness.
    bool is_correct = check_matrix_equality(M, N, C_oracle, C);
    if (is_correct) {
        std::cout << "Tests passed." << std::endl;
    } else {
        std::cout << "Tests failed." << std::endl;
    }

    // Free memory.
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_oracle;
    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);

    // Destroying the cuBLAS handle.
    hipblasDestroy(handle);

    return 0;
}
